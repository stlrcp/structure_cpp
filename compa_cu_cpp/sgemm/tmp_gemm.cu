
#include <hip/hip_runtime.h>
#include <iostream>

void __global__ nativegemm(float *m_a, float *m_b, float *m_c, int m, int n, int k){
    int inx = threadIdx.x;
    // for (int i = 0; i < m; i++){
    //     for (int j = 0; j < k; j++){
    //         float psum = 0.0;
    //         if(inx < n){
    //             psum += m_a[i * n + inx] * m_b[inx * k + j];
    //         }
    //         m_c[i * k + j] = psum;
    //     }   // 写法有大问题，if 只能放在 for 外侧
    // }
    if(inx < m){
        for (int j = 0; j < k; j++){
            float psum = 0.0;
            for (int z = 0; z < n; z++)
                psum += m_a[inx * n + z] * m_b[z * k + j];
            m_c[inx * k + j] = psum;
        }
    }
}

void cgemm(int *h_a, int *h_b, int *h_c, int m, int n, int k){
// void cgemm(float *h_a, float *h_b, float *h_c, int m, int n, int k){
    for (int i = 0; i < m; i++){
        for (int j = 0; j < k; j++){
            float psum = 0.0;
            for (int z = 0; z < n; z++)
                psum += h_a[i * n + z] * h_b[z * k + j];
            h_c[i * k + j] = psum;
        }
    }
}

int main(){
    int *h_a, *h_b, *h_c;
    int m = 3;
    int n = 4;
    int k = 2;
    h_a = (int *)malloc(sizeof(int) * 12);
    h_b = (int *)malloc(sizeof(int) * 8);
    h_c = (int *)malloc(sizeof(int) * 6);

    // float *h_a, *h_b, *h_c;
    // int m = 3;
    // int n = 4;
    // int k = 2;
    // h_a = (float *)malloc(sizeof(int) * m*n);
    // h_b = (float *)malloc(sizeof(int) * n*k);
    // h_c = (float *)malloc(sizeof(int) * m*k);

    float *d_a, *d_b, *d_c;
    size_t size_a = sizeof(float) * m * n;
    size_t size_b = sizeof(float) * n * k;
    size_t size_c = sizeof(float) * m * k;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    for (int i = 0; i < 12; i++)
        h_a[i] = i + 1;
    for (int j = 0; j < 8; j++)
        h_b[j] = j + 1;
    // for (int x = 0; x < 6; x++)
    //     h_c[x] = x;
    cgemm(h_a, h_b, h_c, m, n, k);
    for (int z = 0; z < 6; z++){
        printf("%d ", h_c[z]);     //  类型必须保持一致
        if(z % 2 != 0)
            printf("\n");
    }

    for (int x = 0; x < 6; x++)
        h_c[x] = x;

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);    // 虽然 int 和 float 均占4字节，但类型还是需要保持一致
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    nativegemm<<<1, 5>>>(d_a, d_b, d_c, m, n, k);
    
    hipMemcpy(h_c, d_c, size_c, hipMemcpyDeviceToHost);      //   copy 过程是否存在隐式同步
    hipDeviceSynchronize();       //   后续研究下是否需要
    for (int z = 0; z < 6; z++)
    {
        printf("%d ", h_c[z]);
        if(z % 2 != 0)
            printf("\n");
    }
}
