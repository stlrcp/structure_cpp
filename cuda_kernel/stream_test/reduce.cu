#include "hip/hip_runtime.h"
/*
//  ====== 线程束基本函数与协作组 ======
// 线程束（warp），即一个线程块中连续 32 个线程
// === 单指令-多线程模式 ===
// 一个GPU被分成若干个流多处理器（SM）。核函数中定义的线程块（block）在执行时
// 将被分配到还没有完全占满的 SM。
// 一个 block 不会被分配到不同的 SM，同时一个 SM 中可以有多个 block。不同的
// block 之间可以并发也可以顺序执行，一般不能同步。
// 当某些 block 完成计算任务后，对应的 SM 会部分或完全空闲，然后会有新的
// block 被分配到空闲SM。

// 一个 SM 以32个线程（warp）为单位产生、管理、调度、执行线程。
// 一个 SM 可以处理多个 block，一个 block 可以分为若干个 warp。

// 在同一时刻，一个warp中的线程只能执行一个共同的指令或者闲置，即单指令-多线程执行模型，
// （single instruction multiple thread, SIMT）.

// 当一个线程束中线程顺序的执行判断语句中的不同分支时，称为发生了分支发散（branch divergence）。
if (condition){
    A;
} else {
    B;
}
// 首先，满足 condition 的线程或执行语句A，其他的线程会闲置；
// 然后，不满足条件的将会执行语句B，其他线程闲置。
// 当语句A和B的指令数差不多时，整个 warp 的执行效率就比没有分支的情况 低一半。
// 一般应当在核函数中尽量避免分支发散，但有时这也是不可避免的。
// 如数组计算中常用的判断语句：
if(n<N){
    // do something.
}
// 该分支判断最多影响最后一个 block 中的某些 warp 发生分支发散，一般不会显著地影响性能。
// 有时能通过 合并判断语句 的方式减少分支发散；另外，如果两分支中有一个分支不包含指令，
// 则即使发生分支分散也不会显著影响性能。
// 注意不同架构中的线程调度机制

//  ==== 线程束内的线程同步函数 ====
// __syncwarp(.): 当所涉及的线程都在一个线程束内时，可以将线程块同步函数 __syncthreads() 换成
// 一个更加廉价的线程束同步函数。
// __syncwarp(.): 简称 束内同步函数。
// 函数参数是一个代表掩码的无符号整型数，默认值是全部32个二进制位都为1，代表线程束中
// 的所有线程都参与同步。
// 关于掩码（mask）的简介文章：https://zhuanlan.zhihu.com/p/352025616

// ==== 更多线程束内的基本函数 ====
// == 线程束表决函数 ==
//  - unsgined __ballot_sync(unsigned mask, int predicate),
//    如果线程束内第 n个线程参与计算（旧掩码）且 predicate 值非零，则返回的无符号整型数（新掩码）
//    的第 n 个二进制位为1，否则为0；
//  - int __all_sync(unsigned mask, int predicate),
//    线程束内所有参与线程的 predicate 值均非零，则返回1，否则返回0；
//  - int __any_sync(unsigned mask, int predicate),
//    线程束内所有参与线程的 predicate 值存在非零，则返回1，否则返回0.
// == 线程束洗牌函数：
//  - T __shfl_sync(unsigned mask, T v, int srcLane, int w = warpSize),
//    参与线程返回标号为 srcLane 的线程中变量 v 的值。
//    该函数将一个线程中的数据广播到所有线程
//  - T __shfl_up_sync(unsigned mask, T v, unsigned d, int w=warpSize),
//    标号为t的参与线程返回标号为 t-d 的线程中变量 v 的值， t-d<0 的线程返回 t 线程的变量v。
//    该函数是一种将数据向上平移的操作，即将低线程号的值平移到高线程号。
//    例如当 w=8、d=2时，2-7号线程将返回 0-5线程中变量 v 的值；0-1号线程返回自己的 v。
//  - T __shfl_down_sync(unsigned mask, T v, unsigned d, int w=warpSize),
//    标号为 t 的参与线程返回标号为 t+d 的线程中变量v的值，t+d>w 的线程返回t线程的变量 v。
//    该函数是一种将数据向下平移的操作，即将高线程号的值平移到低线程号。
//    例如当 w=8、d=2时，0-5号线程将返回 2-7 号线程中变量 v 的值，6-7号线程将返回自己的 v。
//  - T __shfl__xor_sync(unsigned mask, T v, int laneMask, int w=warpSize),
//    标号为 t 的参与线程返回标号为 t^laneMask 的线程中变量 v 的值。
//    该函数让线程束内的线程两两交换数据。
// 每个线程束洗牌函数都有一个可选参数 w，默认是线程束大小（32），且只能取2，4，8，16，32.
// 当 w 小于 32 时，相当于逻辑上的线程束大小是 w，其他规则不变。
// 此时，可以定义一个束内索引：（假设使用一维线程块）
int laneId = threadIdx.x % w;   // 线程索引与束内索引的对应关系
// 假设线程块大小为16， w为8：
//   线程索引：0 1 2 3 4 5 6 7 8 9 10 11 12 13 14 15
//   束内索引：0 1 2 3 4 5 6 7 0 1  2  3  4  5  6  7
// 参数中的 mask 称为掩码，是一个无符号整型，具有32位，一般用十六进制表示：
const unsigned FULL_MASK = 0xffffffff;  // '0x'表示十六进制数； '0b'表示二进制数。
// 或者
#define FULL_MASK 0xffffffff
// 以上所有线程束内函数都有 _sync 后缀，表示这些函数都具有 隐式的同步功能。


//  ====  协作组  ====
// 协作组（cooperative groups），可以看作是线程块和线程束同步机制的推广，
// 提供包括线程块内部的同步与协作、线程块之间（网格级）的同步与协作、以及
// 设备与设备之间的同步与协作。
// 使用协作组需要包含如下头文件：
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;
//  === 线程块级别的协作组 ===
// 协作组编程模型中最基本的类型是线程组 thread_group, 其包含如下成员：
//  - void sync(), 同步组内所有线程；
//  - unsigned size(), 返回组内总的线程数目，即组的大小；
//  - unsigned thread_rank(), 返回当前调用该函数的线程在组内的标号（从0计数）；
//  - bool is_valid(), 如果定义的组违反了任何 cuda 限制，返回 false， 否则 true；
// 线程组类型有一个导出类型，线程块 thread_block, 其中定义了额外的函数：
//  - dim3 group_index(), 返回当前调用该函数的线程的线程块指标，等价于 blockIdx；
//  - dim3 thread_index(), 返回当前调用该函数的线程的线程指标，等价于 threadIdx；
// 通过 this_thread_block() 初始化一个线程块对象：
thread_block g = this_thread_block();   // 相当于一个线程块类型的常量。
// 此时：
g.sync() <===>  __syncthreads()
g.group_index() <===> blockIdx
g.thread_index() <===> threadIdx
// 通过 tiled_partition() 可以将一个线程块划分为若干片（tile），每一片构成一个新的线程组。
// 目前，仅支持将片的大小设置为 2 的整数次方且不大于 32.
thread_group g32 = tiled_partition(this_thread_block(), 32);  // 将线程块划分为线程束。
// 可以继续将线程组划分为更细的线程组：
thread_group g4 = tiled_partition(g32, 4);
// 采用模板、在编译期划分 线程块片（thread block tile）:
thread_block_tile<32> g32 = tiled_partition<32>(this_thread_block());
thread_block_tile<32> g4 = tiled_partition<4>(this_thread_block());
// 线程块片具有额外的函数（类似线程束内函数）：
//   unsigned ballot(int predicate);
//   int all(int predicate);
//   int any(int predicate);
//   T shfl(T v, int srcLane);
//   T shfl_up(T v, unsigned d);
//   T shfl_down(T v, unsigned d);
//   T shfl_xor(T v, unsigned d);
// 与一般的线程束不同，线程组内的所有线程都要参与代码运算计算；
// 同时，线程组内函数不需要指定宽度，因为该宽度就是线程块片的大小。

// ===== 数组规约程序的进一步优化 =====
//  === 提高线程利用率 ===
// 在当前的规约程序中，当 offset=64，只用了 1/2 的线程；当 offset=32，只用了 1/4 的线程；...
// 最终，当 offset=1，只用了 1/128 的线程；
// 规约过程一共用了 log2(128) = 7步，平均线程利用率（1/2 + 1/4 + ... + 1/128）/ 7 => 1/7

// 而在规约前的数据拷贝中线程利用率为 100%，可以尽量把计算放在在规约前：让一个线程处理多个数据。

// 一个线程处理相邻若干个数据会导致全局内存的非合并访问。要保证全局内存的合并访问，这里需要
// 保证相邻线程处理相邻数据，一个线程访问的数据需要有某种跨度。
// 该跨度可以是线程块的大小，也可以是网格的大小；对于一维情况，分别是 blockDim.x 和 blockDim.x * gridDim.x

//  === 避免反复分配与释放设备内存 ===
// 设备内存的分配与释放是比较耗时的。
// 通过采用静态全局内存替代动态全局内存，实现编译期的设备内存分配可以更加高效。
// 此外，应当尽量避免在较内存循环反复的分配和释放设备内存
*/


#include "error.cuh"
#include "floats.hpp"
#include "clock.cuh"
#include <hip/hip_cooperative_groups.h>
// #include <iostream>
using namespace cooperative_groups;

__constant__ unsigned FULL_MASK = 0xffffffff;
#define __gSize 10240
__device__ real static_y[__gSize];

__global__ void reduce_syncthreads(real *x, real *y, const int N);
__global__ void reduce_syncwarp(real *x, real *y, const int N);
__global__ void reduce_shfl_down(real *x, real *y, const int N);
__global__ void reduce_cp(real *x, real *y, const int N);
__global__ void reduce_cp_grid(const real *x, real *y, const int N);
real reduce_wrap(const real *x, const int N, const int gSize, const int bSize);
real reduce_wrap_static(const real *x, const int N, const int gSize, const int bSize);

int main(){
    int N = 1e8;
    int M = N * sizeof(real);

    int bSize = 32;
    int gSize = (N + bSize - 1) / bSize;

    cout << FLOAT_PREC << endl;

    real *h_x, *h_x2, *h_y, *h_y2, *h_res;
    h_x = new real[N];
    h_x2 = new real[N];
    h_y = new real[gSize];
    h_y2 = new real[gSize];
    h_res = new real(0.0);
    for (int i = 0; i < N; ++i){
        h_x[i] = 1.23;
        h_x2[i] = 1.23;
    }
    real initRes = 0.0;
    for (int i = 0; i < gSize; ++i){
        h_y2[i] = 0.0;
    }

    cudaClockStart

    real *d_x, *d_y, *d_res;
    CHECK(hipMalloc(&d_x, M));
    CHECK(hipMalloc(&d_y, gSize * sizeof(real)));
    CHECK(hipMalloc(&d_res, sizeof(real)));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyDefault));

    cudaClockCurr

    reduce_syncthreads<<<gSize, bSize, (bSize)*sizeof(real)>>>(d_x, d_y, N);

    CHECK(hipMemcpy(h_y, d_y, gSize * sizeof(real), hipMemcpyDefault));
    real res = 0;
    for (int i = 0; i < gSize; ++i){
        res += h_y[i];
    }
    cout << "reduce_syncthreads result: " << res << endl;
    cudaClockCurr

    CHECK(hipMemcpy(d_res, &initRes, sizeof(real), hipMemcpyDefault));
    reduce_syncwarp<<<gSize, bSize, bSize * sizeof(real)>>>(d_x, d_res, N);
    CHECK(hipMemcpy(h_res, d_res, sizeof(real), hipMemcpyDefault));
    cout << "reduce_syncwrap result: " << *h_res << endl;
    cudaClockCurr

    CHECK(hipMemcpy(d_res, &initRes, sizeof(real), hipMemcpyDefault));
    reduce_shfl_down<<<gSize, bSize, bSize * sizeof(real)>>>(d_x, d_res, N);
    CHECK(hipMemcpy(h_res, d_res, sizeof(real), hipMemcpyDefault));
    cout << "reduce_shfl_down result: " << *h_res << endl;
    cudaClockCurr

    CHECK(hipMemcpy(d_res, &initRes, sizeof(real), hipMemcpyDefault));
    reduce_cp<<<gSize, bSize, bSize * sizeof(real)>>>(d_x, d_res, N);
    CHECK(hipMemcpy(h_res, d_res, sizeof(real), hipMemcpyDefault));
    cout << "reduce_cp result: " << *h_res << endl;
    cudaClockCurr

    reduce_cp_grid<<<gSize, bSize, bSize*sizeof(real)>>>(d_x, d_y, N);
    CHECK(hipMemcpy(h_y, d_y, gSize * sizeof(real), hipMemcpyDefault));
    res = 0.0;
    for (int i = 0; i < gSize; ++i){
        res += h_y[i];
    }
    cout << "reduce_cp_grid result: " << res << endl;
    cudaClockCurr

    res = reduce_wrap(d_x, N, 10240, 128);
    cout << "reduce_wrap result: " << res << endl;
    cudaClockCurr

    res = reduce_wrap_static(d_x, N, 10240, 128);
    cout << "reduce_wrap_static result: " << res << endl;
    cudaClockCurr

    delete[] h_x;
    delete[] h_y;
    delete h_res;
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_res));
    return 0;
}

__global__ void reduce_syncthreads(real *x, real *y, const int N){
    int tid = threadIdx.x;   // 线程块中线程在 x 方向的 id
    int ind = tid + blockIdx.x * blockDim.x;

    extern __shared__ real block_x[];
    block_x[tid] = (ind < N) ? x[ind] : 0;
    __syncthreads();

    for (int offset = blockDim.x / 2; offset > 0; offset /= 2){
        if(tid < offset){
            block_x[tid] += block_x[tid + offset];
        }
        __syncthreads();
    }

    if(tid == 0){
        y[blockIdx.x] = block_x[0];
    }
}

__global__ void reduce_syncwarp(real *x, real *y, const int N){
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int ind = bid * blockDim.x + tid;

    extern __shared__ real block_arr[];
    block_arr[tid] = (ind < N) ? x[ind] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x / 2; offset >= 32; offset /= 2){
        if(tid < offset){
            block_arr[tid] += block_arr[tid + offset];
        }
        __syncthreads();
    }

    for (int offset = 16; offset > 0; offset /= 2){
        if(tid < offset){
            block_arr[tid] += block_arr[tid + offset];
        }
        __syncwarp();
    }

    if(tid == 0){
        atomicAdd(y, block_arr[0]);
    }
}

__global__ void reduce_shfl_down(real *x, real *y, const int N){
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int ind = bid * blockDim.x + tid;

    extern __shared__ real block_arr[];
    block_arr[tid] = (ind < N) ? x[ind] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x / 2; offset >= 32; offset /= 2){
        if(tid < offset){
            block_arr[tid] += block_arr[tid + offset];
        }
        __syncthreads();
    }

    // 在线程寄存器上定义一个变量y
    real curr_y = block_arr[tid];

    for (int offset = 16; offset > 0; offset /= 2){
        curr_y += __shfl_down_sync(FULL_MASK, curr_y, offset);
    }

    if(tid == 0){
        atomicAdd(y, curr_y);
    }
}

__global__ void reduce_cp(real *x, real *y, const int N){
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int ind = bid * blockDim.x + tid;

    extern __shared__ real block_arr[];
    block_arr[tid] = (ind < N) ? x[ind] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x / 2; offset >= 32; offset /= 2){
        if(tid < offset){
            block_arr[tid] += block_arr[tid + offset];
        }
        __syncthreads();
    }

    real curr_y = block_arr[tid];

    thread_block_tile<32> g32 = tiled_partition<32>(this_thread_block());

    for (int offset = 16; offset > 0; offset /= 2){
        curr_y += g32.shfl_down(curr_y, offset);
    }

    if(tid == 0){
        atomicAdd(y, curr_y);
    }
}

__global__ void reduce_cp_grid(const real *x, real *y, const int N){
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    extern __shared__ real block_arr[];

    real curr_y = 0.0;

    const int stride = blockDim.x * gridDim.x;
    for (int n = bid * blockDim.x + tid; n < N; n+= stride){
        curr_y += x[n];
    }

    block_arr[tid] = curr_y;
    __syncthreads();

    for (int offset = blockDim.x / 2; offset >= 32; offset /= 2){
        if(tid < offset){
            block_arr[tid] += block_arr[tid + offset];
        }
        __syncthreads();
    }

    curr_y = block_arr[tid];
    thread_block_tile<32> g32 = tiled_partition<32>(this_thread_block());
    for (int offset = 16; offset > 0; offset /= 2){
        curr_y += g32.shfl_down(curr_y, offset);
    }

    if(tid == 0){
        y[bid] = curr_y;
    }
}

real reduce_wrap(const real *x, const int N, const int gSize, const int bSize){
    const int ymem = gSize * sizeof(real);
    const int smem = bSize * sizeof(real);

    real h_y[1] = {0};
    real *d_y;
    CHECK(hipMalloc(&d_y, ymem));

    reduce_cp_grid<<<gSize, bSize, smem>>>(x, d_y, N);
    reduce_cp_grid<<<1, 1024, 1024 * sizeof(real)>>>(d_y, d_y, gSize);

    CHECK(hipMemcpy(h_y, d_y, sizeof(real), hipMemcpyDefault));
    CHECK(hipFree(d_y));

    return h_y[0];
}

real reduce_wrap_static(const real *x, const int N, const int gSize, const int bSize){
    real *d_y;
    CHECK(hipGetSymbolAddress((void **)&d_y, static_y));
    reduce_cp_grid<<<gSize, bSize, bSize * sizeof(real)>>>(x, d_y, N);
    reduce_cp_grid<<<1, 1024, 1024 * sizeof(real)>>>(d_y, d_y, gSize);

    real h_y[1] = {0};
    CHECK(hipMemcpy(h_y, d_y, sizeof(real), hipMemcpyDefault));

    return h_y[0];
}
